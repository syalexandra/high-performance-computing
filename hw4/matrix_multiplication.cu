#include "hip/hip_runtime.h"
//
//  matrix_multiplication.c
//  
//
//  Created by Yue Sun on 4/18/20.
//

#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

#define N (1UL<<25)
#define BLOCK_SIZE 1024

void vec_inner_product(double* c, const double* a, const double* b){
    double temp=0;
    #pragma omp parallel for shared(temp)
    for (long i = 0; i < N; i++) {
        temp += a[i] * b[i];
    }
    
    *c=temp;
    
}



__global__
void vec_inner_product_kernel(double* c,const double* a, const double* b){

    __shared__ float chache[BLOCK_SIZE] ;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int chacheindex = threadIdx.x;
    double temp;
    
    while ( tid < N )
    {
         temp += a[tid] * b[tid] ;
         tid += blockDim.x * gridDim.x ;
    }
    
    chache[chacheindex] = temp ;

    __synchthreads ();
    int i  = blockDim.x / 2;
    while ( i!=0 )
     {
        if ( chacheindex < i )
            chache[chacheindex] += chache [chacheindex + i] ;
        
        __synchthreads();
        i/=2 ;
     }
     
     if ( chacheindex == 0 )c[blockIdx.x] = chache [0] ;

    
}




void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {

    double *x, *y, *z, *s;
    hipMallocManaged(&x, N * sizeof(double));
    hipMallocManaged(&y, N * sizeof(double));
    hipMallocManaged(&s, BLOCK_SIZE * sizeof(double));
    
    //hipMallocManaged(&s, N * sizeof(double));
    
    double* s_ref;
    
    
    #pragma omp parallel for schedule(static)
    for (long i = 0; i < N; i++)
    {
      x[i] = i+2;
      y[i] = 1.0/(i+1);
    }
    
    
    
    double tt = omp_get_wtime();
    vec_inner_product(s_ref,x, y);
    printf("CPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
    

    tt = omp_get_wtime();
    vec_inner_product_kernel<<<N/1024+1,1024>>>(s,x,y);
    hipDeviceSynchronize();
    
    float sum = 0;
    for ( int i = 0 ; i<BLOCK_SIZE ; i++ )
        sum+=s[i];
    printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
    
    
    double err = 0;
    //for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
    err=s_ref-sum;
    printf("Error = %f %f %f\n", err,s_ref,sum);
    
    
    
}
