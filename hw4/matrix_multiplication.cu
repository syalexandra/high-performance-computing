#include "hip/hip_runtime.h"
//
//  matrix_multiplication.c
//  
//
//  Created by Yue Sun on 4/18/20.
//

#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

#define N (1UL<<25);
void vec_inner_product(double* c, const double* a, const double* b){
    double temp=0;
    #pragma omp parallel for schedule(static) shared(temp)
    for (long i = 0; i < N; i++) {
        temp += a[i] * b[i];
    }
    *c=temp;
    
}



__global__
void vec_inner_product_kernel(double* c, const double* a, const double* b){
    __shared__ double temp[N];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) temp[idx] = a[idx] * b[idx];
    
    __syncthreads();
    if(0==idx){
        int sum=0;
        for(int i=0;i<N;i++){
            sum+=temp[i];
        }
        *c=sum;
    }
    
}




void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {

    double *x, *y, *z;
    hipMallocManaged(&x, N * sizeof(double));
    hipMallocManaged(&y, N * sizeof(double));
    hipMallocManaged(&z, sizeof(double));
    
    double* z_ref = (double*) malloc(sizeof(double));
    #pragma omp parallel for schedule(static)
    for (long i = 0; i < N; i++)
    {
      x[i] = i+2;
      y[i] = 1.0/(i+1);
    }
    
    z[0]=0;
    z_ref[0]=0;
    
    double tt = omp_get_wtime();
    vec_inner_product(z_ref, x, y);
    printf("CPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

    tt = omp_get_wtime();
    vec_inner_product_kernel<<<N/1024+1,1024>>>(z, x, y);
    hipDeviceSynchronize();
    printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
    
    
    double err = 0;
    for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
    printf("Error = %f\n", err);
}
