//
//  matrix_multiplication.c
//  
//
//  Created by Yue Sun on 4/18/20.
//


#include <hip/hip_runtime.h>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

#define N (1UL<<25)


void vec_inner_product(double* c, const double* a, const double* b){
    double temp=0;
    #pragma omp parallel for shared(temp)
    for (long i = 0; i < N; i++) {
        temp += a[i] * b[i];
    }
    
    *c=temp;
    
}



__global__
void vec_inner_product_kernel(double* c, double *sum,const double* a, const double* b){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) c[idx] = a[idx] * b[idx];
    
    __syncthreads();
    if(0==idx){
        for(int i=0;i<N;i++){
            *sum+=c[i];
        }
    }
    
}




void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

int main() {

    double *x, *y, *z,*s;
    hipMallocManaged(&x, N * sizeof(double));
    hipMallocManaged(&y, N * sizeof(double));
    hipMallocManaged(&z, N * sizeof(double));
    hipMallocManaged(&s, sizeof(double));
    
    double* s_ref = (double*) malloc(sizeof(double));
    
    
    #pragma omp parallel for schedule(static)
    for (long i = 0; i < N; i++)
    {
      x[i] = i+2;
      y[i] = 1.0/(i+1);
      z[i] = 0;
    }
    s_ref[0]=0;
    s[0]=0;
    
    double tt = omp_get_wtime();
    vec_inner_product(s_ref,x, y);
    printf("CPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

    tt = omp_get_wtime();
    vec_inner_product_kernel<<<N/1024+1,1024>>>(z,s, x, y);
    
    //cudaDeviceSynchronize();
    printf("GPU Bandwidth = %f GB/s\n", 3*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
    
    
    double err = 0;
    //for (long i = 0; i < N; i++) err += fabs(z[i]-z_ref[i]);
    err=s_ref[0]-s[0];
    printf("Error = %f\n", err);
}
