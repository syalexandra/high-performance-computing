#include "hip/hip_runtime.h"
//
//  hogwild.cpp
//  parallelSGD
//
//  Created by Yue Sun on 4/22/20.
//  Copyright © 2020 Yue Sun. All rights reserved.
//


#include "dataReader.h"


#include <iostream>
#include <cstdlib>
#include <string>
#include <random>
#include <chrono>
#include <omp.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

using namespace std;
typedef unsigned char uchar;

__host__ __device__ double getOneGradient(double* weight,int index,const double*trainingData,const uchar* trainingLabel,double eta,int n_data,int n_weights,int n_labels,double lambda){
    
    double delta_weight=0;
    int i=index / n_weights;//i is for label i
    int j=index % n_weights;//j is for data j
    //printf("%d %d %d %d %d %d \n",index,i,j,n_weights,n_data,n_labels);
    
    double* probList;
    probList=(double*)malloc(n_labels*sizeof(double));
    for(int b=0;b<n_data;b++){
    
        double probSum=0;
        
        
        for(int l=0;l<n_labels;l++){
            double probExp=0;
            for(int w=0;w<n_weights;w++){
                //printf("%d %d %d %f %f \n",l,w,b,weight[l*n_weights+w],trainingData[b*n_weights+w]);
                probExp+=weight[l*n_weights+w]*trainingData[b*n_weights+w];
            }
            //printf("%d, %d, probExp %f ",i,j,exp(probExp));
            probList[l]=exp(probExp);
            probSum+=exp(probExp);
            printf("%d,%d,%d\n",index,l);
        }
        
        printf("probList[i]: %f \n",probList[i]);
        probList[i]/=probSum;
        
        
        double sign = (trainingLabel[b]==i)?1:0;
        double partialDerivative = (sign-probList[i])*trainingData[b*n_weights+j];
        partialDerivative += lambda * 2 * weight[i*n_weights+j];
        delta_weight -= partialDerivative;
        
    }
    printf("index: %d delta_weight: %f\n",index,delta_weight);
    free(probList);
    return delta_weight;
    
}

__global__ void updateWeightKernel(double* weight,const double* trainingData,const uchar* trainingLabel,double eta,int n_data,int n_weights,int n_labels,int batchSize,double lambda){
    
    int x=blockIdx.x*blockDim.x+threadIdx.x;
    int y=blockIdx.y*blockDim.y+threadIdx.y;
    int index=x*gridDim.x*blockDim.x+y;
    //printf("index %d \n",index);
    int weight_size=n_weights*n_labels;
    
    if(index<weight_size){
        double deltaWeight;
        double* data;
        data=(double*)malloc(batchSize*n_weights*sizeof(double));
        
        uchar* label;
        label=(uchar*)malloc(batchSize*sizeof(uchar));
        
        for(int b=0;b<batchSize;b++){
            hiprandState_t state;
            hiprand_init(index,0,b,&state);
            int r;
            r=hiprand(&state)%n_data;
            label[b]=trainingLabel[r];
            //printf("%d ",label[b]);
            for(int w=0;w<n_weights;w++){
                data[b*n_weights+w]=trainingData[r*n_weights+w];
                //printf("%d %d %f \n",r,w,trainingData[r*n_weights+w]);
            }
        }
        
        deltaWeight=getOneGradient(weight,index, data, label,eta, batchSize, n_weights, n_labels,lambda/batchSize);
        weight[index]-=eta* deltaWeight;
        free(data);
        free(label);
    }
    
}



int main(int argc, const char * argv[]) {
    // insert code here...
    mnist data;
    int n_images;
    int size_image;
    double **tempData;
    //trainingData = data.read_mnist_images("train-images-idx3-ubyte",n_images, size_image);
    tempData = data.read_mnist_images("train-images.idx3-ubyte",n_images, size_image);
    //n_images 60000,size_images=785
    double * trainingData;
    hipHostMalloc((void**)&trainingData,n_images*(size_image+1)*sizeof(double));
    
    for(int i=0;i<n_images;i++){
        for(int j=0;j<size_image+1;j++){
            trainingData[i*(size_image+1)+j]=tempData[i][j];
        }
    }
    
    int n_labels;
    uchar *tempLabel;
    
    tempLabel = data.read_mnist_labels("train-labels.idx1-ubyte",n_labels);
    uchar* trainingLabel;
    hipHostMalloc((void**)&trainingLabel,n_labels*sizeof(uchar));
    for(int i=0;i<n_labels;i++){
        trainingLabel[i]=tempLabel[i];
    }
    
    dim3 gridSize(4,4);
    dim3 blockSize(10,10);
    
    
    double* weight;
    int weight_size=(size_image+1)*10;
    hipHostMalloc((void**)&weight,weight_size*sizeof(double));
    
    //initialize the weight
    int seed =1;//chrono::system_clock::now().time_since_epoch().count();
    default_random_engine generator (seed);
    normal_distribution<double> distribution (0.0,1.0);
    for (int i=0;i<weight_size;i++){
        weight[i]=distribution(generator);
    }
    
    printf("Enter iterations (> 10):\n");
    int n_iterations;
    //scanf("%d", &n_iterations);
    n_iterations=10;
    double eta;
    eta=0.001;
    printf("\nEnter learning rate (eta = 0.001):\n");
    //scanf("%lf", &eta);
    
    double lambda;
    lambda=0.001;
    printf("\nEnter regularization parameter (lambda = 0.001):\n");
    
    //printf("%d %d \n",gridSize,blockSize);
    //update the weight
    for(int j=0;j<n_iterations;j++){
        
        updateWeightKernel<<<gridSize,blockSize>>>(weight,trainingData,trainingLabel,eta,n_images,size_image+1,10,2,lambda);
        hipDeviceSynchronize();
        
    }
    
    free(tempData);
    free(tempLabel);
    hipFree(trainingData);
    hipFree(trainingLabel);
    hipFree(weight);
    
    
    
    
}
