#include "hip/hip_runtime.h"
//
//  hogwild.cpp
//  parallelSGD
//
//  Created by Yue Sun on 4/22/20.
//  Copyright © 2020 Yue Sun. All rights reserved.
//


#include "dataReader.h"


#include <iostream>
#include <cstdlib>
#include <string>
#include <random>
#include <chrono>
#include <omp.h>
#include <vector>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

using namespace std;
typedef unsigned char uchar;

__host__ __device__ double getOneGradient(double* weight,int index,const double*trainingData,const uchar* trainingLabel,double eta,int n_data,int n_weights,int n_labels,double lambda){
    
    double delta_weight=0;
    int i=index / n_weights;//i is for label i
    int j=index % n_weights;//j is for data j
    //printf("%d %d %d \n",i,j,n_data);
    
    for(int b=0;b<n_data;b++){
    
        double probSum=0;
        double* probList;
        probList=(double*)malloc(n_labels*sizeof(double));
        
        for(int l=0;l<n_labels;l++){
            double probExp=0;
            for(int w=0;w<n_weights;w++){
                //printf("%f %f \n",weight[l*n_weights+w],trainingData[b*n_weights+w]);
                probExp+=weight[l*n_weights+w]*trainingData[b*n_weights+w];
            }
            //printf("%f ",probExp);
            probList[l]=exp(probExp);
            probSum+=exp(probExp);
        }
        
        //printf("%f ",probSum);
        probList[i]/=probSum;
        
        
        double sign = (trainingLabel[b]==i)?1:0;
        double partialDerivative = (sign-probList[i])*trainingData[b*n_weights+j];
        partialDerivative += lambda * 2 * weight[i*n_weights+j];
        delta_weight -= partialDerivative;
        
    }
    //printf("delta_weight %f",delta_weight);
    return delta_weight;
    
}

__global__ void updateWeightKernel(double* weight,const double* trainingData,const uchar* trainingLabel,double eta,int n_data,int n_weights,int n_labels,int batchSize,double lambda){
    
    int index=blockIdx.x*blockDim.x+threadIdx.x;
    int weight_size=n_weights*n_labels;
    
    
    if(index<weight_size){
        double deltaWeight;
        double* data;
        //hipMalloc(&data,batchSize*n_weights*sizeof(double));
        data=(double*)malloc(batchSize*n_weights*sizeof(double));
        
        uchar* label;
        //hipMalloc(&label,batchSize*sizeof(uchar));
        label=(uchar*)malloc(batchSize*sizeof(uchar));
        
        for(int b=0;b<batchSize;b++){
            hiprandState_t state;
            hiprand_init(index,0,b,&state);
            int r;
            r=hiprand(&state)%n_data;
            //hipMemcpy(data,trainingData+r*n_weights,n_weights*sizeof(double),hipMemcpyHostToDevice);
            //hipMemcpy(label,trainingLabel+r,sizeof(uchar),hipMemcpyHostToDevice);
            label[b]=trainingLabel[r];
            //printf("%d ",label[b]);
            for(int w=0;w<n_weights;w++){
                data[b*n_weights+w]=trainingData[r*n_weights+w];
                printf("%f ",data[b*n_weights+w]);
            }
        }
        
        deltaWeight=getOneGradient(weight,index, data, label,eta, batchSize, n_weights, n_labels,lambda/batchSize);
        weight[index]-=eta* deltaWeight;
    }
    
}



int main(int argc, const char * argv[]) {
    // insert code here...
    mnist data;
    int n_images;
    int size_image;
    double **tempData;
    //trainingData = data.read_mnist_images("train-images-idx3-ubyte",n_images, size_image);
    tempData = data.read_mnist_images("train-images.idx3-ubyte",n_images, size_image);
    //n_images 60000,size_images=785
    double * trainingData;
    hipHostMalloc((void**)&trainingData,n_images*(size_image+1)*sizeof(double));
    
    for(int i=0;i<n_images;i++){
        for(int j=0;j<size_image+1;j++){
            trainingData[i*(size_image+1)+j]=tempData[i][j];
        }
    }
    
    int n_labels;
    uchar *tempLabel;
    
    tempLabel = data.read_mnist_labels("train-labels.idx1-ubyte",n_labels);
    uchar* trainingLabel;
    hipHostMalloc((void**)&trainingLabel,n_labels*sizeof(uchar));
    for(int i=0;i<n_labels;i++){
        trainingLabel[i]=tempLabel[i];
    }
    
    int gridSize=size_image+1;
    int blockSize=2;
    
    
    double* weight;
    int weight_size=(size_image+1)*10;
    hipHostMalloc((void**)&weight,weight_size*sizeof(double));
    
    //initialize the weight
    int seed =1;//chrono::system_clock::now().time_since_epoch().count();
    default_random_engine generator (seed);
    normal_distribution<double> distribution (0.0,1.0);
    for (int i=0;i<weight_size;i++){
        weight[i]=distribution(generator);
    }
    
    printf("Enter iterations (> 10):\n");
    int n_iterations;
    //scanf("%d", &n_iterations);
    n_iterations=10;
    double eta;
    eta=0.001;
    printf("\nEnter learning rate (eta = 0.001):\n");
    //scanf("%lf", &eta);
    
    double lambda;
    lambda=0.001;
    printf("\nEnter regularization parameter (lambda = 0.001):\n");
    
    //update the weight
    for(int j=0;j<n_iterations;j++){
        
        updateWeightKernel<<<gridSize,blockSize>>>(weight,trainingData,trainingLabel,eta,n_images,size_image+1,10,2,lambda);
        hipDeviceSynchronize();
        
    }
    
    
    
    
}
